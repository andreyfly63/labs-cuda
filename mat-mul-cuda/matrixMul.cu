#include "hip/hip_runtime.h"
/////////////////////////////////////////////////////////////////
//matrix multiplication is performed in this lab               //
//                                                             //
//for run:                                                     //
//1) make                                                      //
//2) ./makefile                                                //
//                                                             //
/////////////////////////////////////////////////////////////////


#include <iostream>
#include <vector>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "kernel.h"
#include "kernel.cu"
#include "dev_array.h"
#include <math.h>

using namespace std;

int main()
{
    // Perform matrix multiplication C = A*B
    // where A, B and C are NxN matrices
    int N = 122;
    int SIZE = N*N;

    // Allocate memory on the host
    vector<float> h_A(SIZE);
    vector<float> h_B(SIZE);
    vector<float> h_C(SIZE);

    // Initialize matrices on the host
    for (int i=0; i<N; i++){
        for (int j=0; j<N; j++){
            h_A[i*N+j] = i+i*j;
            h_B[i*N+j] = i-i*j);
        }
    }
    // Print elements matrix
    for (int  i=0; i<N*N; i++){
    	cout<<h_A[i]<<endl;
    }
    // Allocate memory on the device
    dev_array<float> d_A(SIZE);
    dev_array<float> d_B(SIZE);
    dev_array<float> d_C(SIZE);

    d_A.set(&h_A[0], SIZE);
    d_B.set(&h_B[0], SIZE);
    clock_t start, end;
    start = clock();
    matrixMultiplication(d_A.getData(), d_B.getData(), d_C.getData(), N);
    hipDeviceSynchronize();

    d_C.get(&h_C[0], SIZE);
    hipDeviceSynchronize();
    end = clock();
    cout<<"time for matrixMul in GPU: "<< end-start<<endl;
    float *cpu_C;
    cpu_C=new float[SIZE];
    start = clock();
    // Now do the matrix multiplication on the CPU
    float sum;
    for (int row=0; row<N; row++){
        for (int col=0; col<N; col++){
            sum = 0.f;
            for (int n=0; n<N; n++){
                sum += h_A[row*N+n]*h_B[n*N+col];
            }
            cpu_C[row*N+col] = sum;
        }
    }
    end = clock();
    cout<<"time for matrixMul in CPU: "<< end-start<<endl;
    cout << "cpu_C: "<< cpu_C << endl;
    return 0;
}

